#include "hip/hip_runtime.h"
#include "../ops.hpp"
#include "ops_common.cuh"
#include "artifacts.hpp"
#include "flashinfer/sampling.cuh"
#include "../test_kernels/test_kernels.cuh"
#include <random>
#include <sstream>
#include <vector>
#include <optional>
#include <type_traits>

namespace ops {

template<typename T>
void run_softmax_typed(const std::string& case_id, const SoftmaxConfig& cfg, uint64_t seed) {
	const int batch_size = cfg.batch_size;
	const int vocab_size = cfg.vocab_size;
	const float temperature = cfg.temperature;

	std::mt19937_64 rng(seed);
	std::uniform_real_distribution<float> dist(-5.0f, 5.0f);  // Reasonable logit range

	// Input logits [batch_size, vocab_size]
	const size_t logits_size = static_cast<size_t>(batch_size) * vocab_size;
	std::vector<T> h_input_logits(logits_size);
	std::vector<T> h_output(logits_size, 0);

	for (auto& v : h_input_logits) v = f2t<T>(dist(rng));

	// Device allocation
	T* d_input_logits = nullptr;
	T* d_output = nullptr;
	T* d_temperature_arr = nullptr;  // Per-batch temperature (can be nullptr for scalar temp)
	void* d_workspace = nullptr;

	check_cuda(hipMalloc(&d_input_logits, logits_size * sizeof(T)));
	check_cuda(hipMalloc(&d_output, logits_size * sizeof(T)));

	// Calculate workspace size (estimate)
	const size_t workspace_size = batch_size * vocab_size * sizeof(T);
	check_cuda(hipMalloc(&d_workspace, workspace_size));

	hipStream_t stream;
	check_cuda(hipStreamCreate(&stream));

	// Copy to device
	check_cuda(hipMemcpyAsync(d_input_logits, h_input_logits.data(), logits_size * sizeof(T), hipMemcpyHostToDevice, stream));

	// Apply FlashInfer OnlineSoftmax - use test-local wrapper for non-float types
	hipError_t softmax_result;
	if constexpr (std::is_same_v<T, float>) {
		// Use FlashInfer directly for float (same as backend)
		softmax_result = flashinfer::sampling::OnlineSoftmax<T>(
			d_input_logits,
			d_output,
			batch_size,
			vocab_size,
			d_temperature_arr,  // nullptr for scalar temperature
			f2t<T>(temperature),
			d_workspace,
			workspace_size,
			false,  // enable_pdl
			stream
		);
	} else {
		// Use test-local wrapper for other dtypes
		softmax_result = online_softmax_test_local<T>(
			d_input_logits,
			d_output,
			batch_size,
			vocab_size,
			d_temperature_arr,  // nullptr for scalar temperature
			f2t<T>(temperature),
			d_workspace,
			workspace_size,
			false,  // enable_pdl
			stream
		);
	}

	if (softmax_result != hipSuccess) {
		std::cerr << "FlashInfer OnlineSoftmax failed: " << hipGetErrorString(softmax_result) << std::endl;
		// Fallback: just copy input to output for testing purposes
		check_cuda(hipMemcpyAsync(d_output, d_input_logits, logits_size * sizeof(T), hipMemcpyDeviceToDevice, stream));
	}

	check_cuda(hipStreamSynchronize(stream));

	// Generate dtype string and case_id
	std::string dtype_name;
	if constexpr (std::is_same_v<T, float>) dtype_name = "fp32";
	else if constexpr (std::is_same_v<T, __half>) dtype_name = "fp16";
	else if constexpr (std::is_same_v<T, __hip_bfloat16>) dtype_name = "bf16";

	// Write artifacts
	if (artifacts::op_enabled("softmax")) {
		auto dir = artifacts::ensure_dir_for_case("softmax", case_id + "_" + dtype_name);

		artifacts::write_device_bin(dir, "input_logits", d_input_logits, logits_size);
		artifacts::write_device_bin(dir, "output", d_output, logits_size);

		std::ostringstream meta;
		meta << "\"version\": \"1\",\n"
			 << "\"op\": \"softmax\",\n"
			 << "\"case_id\": " << artifacts::json_escape(case_id + "_" + dtype_name) << ",\n"
			 << "\"config\": {\"batch_size\": " << batch_size
			 << ", \"vocab_size\": " << vocab_size
			 << ", \"temperature\": " << temperature << "},\n"
			 << "\"dtype_map\": {\"input_logits\": \"" << dtype_name << "\", \"output\": \"" << dtype_name << "\"},\n"
			 << "\"shape_map\": {\"input_logits\": [" << batch_size << ", " << vocab_size
			 << "], \"output\": [" << batch_size << ", " << vocab_size << "]}";
		artifacts::write_meta_json(dir, meta.str());
	}

	// Cleanup
	hipStreamDestroy(stream);
	hipFree(d_workspace);
	hipFree(d_output);
	hipFree(d_input_logits);
}

void run_softmax(const std::string& case_id,
				 const SoftmaxConfig& cfg,
				 uint64_t seed) {
	using T = float;  // FlashInfer OnlineSoftmax supports float

	const int batch_size = cfg.batch_size;
	const int vocab_size = cfg.vocab_size;
	const float temperature = cfg.temperature;

	std::mt19937_64 rng(seed);
	std::uniform_real_distribution<float> dist(-5.0f, 5.0f);  // Reasonable logit range

	// Input logits [batch_size, vocab_size]
	const size_t logits_size = static_cast<size_t>(batch_size) * vocab_size;
	std::vector<T> h_input_logits(logits_size);
	std::vector<T> h_output(logits_size, 0);

	for (auto& v : h_input_logits) v = static_cast<T>(dist(rng));

	// Device allocation
	T* d_input_logits = nullptr;
	T* d_output = nullptr;
	T* d_temperature_arr = nullptr;  // Per-batch temperature (can be nullptr for scalar temp)
	void* d_workspace = nullptr;

	check_cuda(hipMalloc(&d_input_logits, logits_size * sizeof(T)));
	check_cuda(hipMalloc(&d_output, logits_size * sizeof(T)));

	// Calculate workspace size (estimate)
	const size_t workspace_size = batch_size * vocab_size * sizeof(T);
	check_cuda(hipMalloc(&d_workspace, workspace_size));

	hipStream_t stream;
	check_cuda(hipStreamCreate(&stream));

	// Copy to device
	check_cuda(hipMemcpyAsync(d_input_logits, h_input_logits.data(), logits_size * sizeof(T), hipMemcpyHostToDevice, stream));

	// Apply FlashInfer OnlineSoftmax
	hipError_t softmax_result = flashinfer::sampling::OnlineSoftmax<T>(
		d_input_logits,
		d_output,
		batch_size,
		vocab_size,
		d_temperature_arr,  // nullptr for scalar temperature
		temperature,        // scalar temperature value
		d_workspace,
		workspace_size,
		false,  // enable_pdl
		stream
	);

	if (softmax_result != hipSuccess) {
		std::cerr << "FlashInfer OnlineSoftmax failed: " << hipGetErrorString(softmax_result) << std::endl;
		// Fallback: just copy input to output for testing purposes
		check_cuda(hipMemcpyAsync(d_output, d_input_logits, logits_size * sizeof(T), hipMemcpyDeviceToDevice, stream));
	}

	check_cuda(hipStreamSynchronize(stream));

	// Write artifacts
	if (artifacts::op_enabled("softmax")) {
		auto dir = artifacts::ensure_dir_for_case("softmax", case_id);

		artifacts::write_device_bin(dir, "input_logits", d_input_logits, logits_size);
		artifacts::write_device_bin(dir, "output", d_output, logits_size);

		std::ostringstream meta;
		meta << "\"version\": \"1\",\n"
			 << "\"op\": \"softmax\",\n"
			 << "\"case_id\": " << artifacts::json_escape(case_id) << ",\n"
			 << "\"config\": {\"batch_size\": " << batch_size
			 << ", \"vocab_size\": " << vocab_size
			 << ", \"temperature\": " << temperature << "},\n"
			 << "\"dtype_map\": {\"input_logits\": \"fp32\", \"output\": \"fp32\"},\n"
			 << "\"shape_map\": {\"input_logits\": [" << batch_size << ", " << vocab_size
			 << "], \"output\": [" << batch_size << ", " << vocab_size << "]}";
		artifacts::write_meta_json(dir, meta.str());
	}

	// Cleanup
	hipStreamDestroy(stream);
	hipFree(d_workspace);
	hipFree(d_temperature_arr);
	hipFree(d_output);
	hipFree(d_input_logits);
}

// Note: FlashInfer OnlineSoftmax has internal float assumptions
// Only instantiate float for now - __half and __hip_bfloat16 have compilation issues
template void run_softmax_typed<float>(const std::string& case_id, const SoftmaxConfig& cfg, uint64_t seed);

} // namespace ops
