#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblaslt.h>
#include <hipblas.h>
#include "common.cuh"

// Test-local implementation of GEMM for native fp16 support
// This uses cuBLASLt with HIPBLAS_COMPUTE_16F for native fp16 computation

template<>
void gemm_cublasLt<__half>(hipblasLtHandle_t ltHandle, hipStream_t stream,
                          const __half* d_A, const __half* d_B, const __half* d_bias, __half* d_C,
                          int m, int n, int k, void* d_workspace, size_t workspaceSize,
                          bool transa, bool transb) {
    
    hipblasLtMatmulDesc_t matmulDesc;
    hipblasLtMatrixLayout_t Adesc, Bdesc, Cdesc, biasDesc;
    
    // Create matrix multiplication descriptor with native fp16 compute
    hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_16F, HIP_R_16F);
    
    // Set transpose operations
    hipblasOperation_t op_A = transa ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t op_B = transb ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &op_A, sizeof(op_A));
    hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &op_B, sizeof(op_B));
    
    // Set bias pointer if provided
    if (d_bias) {
        hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &d_bias, sizeof(d_bias));
    }
    
    // Create matrix layout descriptors
    int lda = transa ? m : k;
    int ldb = transb ? k : n;
    int ldc = n;
    
    hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_16F, transa ? k : m, transa ? m : k, lda);
    hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_16F, transb ? n : k, transb ? k : n, ldb);
    hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_16F, m, n, ldc);
    
    if (d_bias) {
        hipblasLtMatrixLayoutCreate(&biasDesc, HIP_R_16F, n, 1, n);
    }
    
    // Set scaling factors (alpha=1, beta=0)
    __half alpha = __float2half(1.0f);
    __half beta = __float2half(0.0f);
    
    // Find the best algorithm
    hipblasLtMatmulPreference_t preference;
    hipblasLtMatmulPreferenceCreate(&preference);
    hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                                        &workspaceSize, sizeof(workspaceSize));
    
    // Get heuristic
    hipblasLtMatmulHeuristicResult_t heuristic;
    int returnedResults = 0;
    hipblasLtMatmulAlgoGetHeuristic(ltHandle, matmulDesc, Adesc, Bdesc, Cdesc, Cdesc,
                                  preference, 1, &heuristic, &returnedResults);
    
    // Perform matrix multiplication
    if (returnedResults > 0) {
        hipblasLtMatmul(ltHandle, matmulDesc,
                      &alpha, d_A, Adesc, d_B, Bdesc,
                      &beta, d_C, Cdesc, d_C, Cdesc,
                      &heuristic.algo, d_workspace, workspaceSize, stream);
    }
    
    // Clean up
    hipblasLtMatrixLayoutDestroy(Adesc);
    hipblasLtMatrixLayoutDestroy(Bdesc);
    hipblasLtMatrixLayoutDestroy(Cdesc);
    if (d_bias) hipblasLtMatrixLayoutDestroy(biasDesc);
    hipblasLtMatmulPreferenceDestroy(preference);
    hipblasLtMatmulDescDestroy(matmulDesc);
}