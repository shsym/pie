#include "hip/hip_runtime.h"
#include "l4ma.cuh"
#include "config.hpp"
#include "common.cuh"   // Your helper functions header
#include "stack_allocator.cuh" // Import the new stack allocator

#include <stdexcept>
#include <iostream>
#include <utility>
#include <algorithm> // for std::max

#include "flashinfer/norm.cuh"
#include "flashinfer/pos_enc.cuh"
#include "flashinfer/page.cuh"
#include "flashinfer/activation.cuh"
#include "flashinfer/sampling.cuh"
#include "flashinfer/vec_dtypes.cuh"

#include "flashinfer_ops.cuh"
#include "kernels.cuh"  // extracted primitive kernels & launchers
#include "artifacts.hpp" // optional artifact dump utilities

std::vector<uint8_t> packbits_little(const std::vector<bool>& data) {
    // Calculate the number of bytes needed, padding with zeros for the last byte if necessary.
    const size_t num_bytes = (data.size() + 7) / 8;
    std::vector<uint8_t> packed(num_bytes, 0);

    for (size_t i = 0; i < data.size(); ++i) {
        // The first element in each chunk of 8 corresponds to the LSB.
        // The '& 7' is equivalent to 'i % 8' but can be faster.
        if (data[i]) {
            packed[i / 8] |= (1 << (i & 7));
        }
    }

    return packed;
}


template <typename T>
L4maBuffer<T>::L4maBuffer(const L4maConfig& cfg, int32_t page_size, int32_t dist_size, size_t workspace_size)
    : config(cfg),
      page_size(page_size),
      dist_size(dist_size),
      num_tokens(0),
      batch_size(0),
      stream(nullptr),
      buffer_size_(workspace_size) {
    allocator_ = std::make_unique<StackAllocator>(buffer_size_);
    CUBLAS_CHECK(hipblasLtCreate(&ltHandle));
}

// destructor
template <typename T>
L4maBuffer<T>::~L4maBuffer() {
    // Clean up the CUBLAS handle
    CUBLAS_CHECK(hipblasLtDestroy(ltHandle));
    // The StackAllocator will automatically free its buffer when it goes out of scope
}

template <typename T>
size_t L4maBuffer<T>::get_workspace_size(
    const L4maConfig& config,
    size_t max_num_tokens,
    size_t max_batch_size,
    size_t max_kv_seqlens, // TODO: max_dist_size is needed for sampling buffers
    size_t dist_size
) {
    const size_t alignment = 256;
    const size_t hidden_size = config.hidden_size;
    const size_t intermediate_size = config.intermediate_size;
    const size_t num_q_heads = config.num_query_heads;
    const size_t num_kv_heads = config.num_key_value_heads;
    const size_t head_size = config.head_size;

    // --- Peak memory within a decoder layer ---
    size_t decoder_layer_peak = 0;
    {
        // Buffers allocated in L4maDecoderLayer::forward
        size_t decoder_wrapper_bytes = 2 * align_up((size_t)max_num_tokens * hidden_size * sizeof(T), alignment);

        // Path 1: Attention block peak
        size_t attn_path_peak = 0;
        attn_path_peak += align_up((size_t)max_num_tokens * num_q_heads * head_size * sizeof(T), alignment); // q_proj
        attn_path_peak += align_up((size_t)max_num_tokens * num_kv_heads * head_size * sizeof(T), alignment); // k_proj
        attn_path_peak += align_up((size_t)max_num_tokens * num_kv_heads * head_size * sizeof(T), alignment); // v_proj
        attn_path_peak += align_up(32 * 1024 * 1024, alignment); // cublas_workspace

        // Path 2: MLP block peak
        size_t mlp_path_peak = 0;
        mlp_path_peak += align_up((size_t)max_num_tokens * intermediate_size * sizeof(T), alignment); // up_proj
        mlp_path_peak += align_up((size_t)max_num_tokens * intermediate_size * sizeof(T), alignment); // gate_proj
        mlp_path_peak += align_up(32 * 1024 * 1024, alignment); // cublas_workspace

        decoder_layer_peak = decoder_wrapper_bytes + std::max(attn_path_peak, mlp_path_peak);
    }

    // --- Peak memory for the final LM head and sampling ---
    size_t final_stage_peak = 0;
    {
        final_stage_peak += align_up((size_t)max_num_tokens * hidden_size * sizeof(T), alignment);                      // hidden_states
        final_stage_peak += align_up((size_t)max_num_tokens * config.vocab_size * sizeof(T), alignment);                // output_logits
        final_stage_peak += align_up((size_t)max_num_tokens * config.vocab_size * sizeof(float), alignment);             // output_logits_fp32
        final_stage_peak += align_up((size_t)max_num_tokens * config.vocab_size * sizeof(float), alignment);             // output_logits_masked
        final_stage_peak += align_up((size_t)max_num_tokens * dist_size * sizeof(float), alignment);                     // final_logits_val
        final_stage_peak += align_up((size_t)max_num_tokens * dist_size * sizeof(int32_t), alignment);                   // final_logits_indices
        final_stage_peak += align_up(32 * 1024 * 1024, alignment);                                                      // lm_head_workspace
        final_stage_peak += align_up((size_t)max_num_tokens * hidden_size * sizeof(T), alignment);                      // gathered_states (worst case)
    }

    // --- Other persistent buffers ---
    size_t persistent_buffers = 0;
    // Memory for FlashInfer handlers
    persistent_buffers += align_up(256 * 1024 * 1024, alignment);
    persistent_buffers += align_up(8 * 1024 * 1024, alignment);
    // Working buffer in L4maModel::forward
    persistent_buffers += align_up((size_t)max_num_tokens * hidden_size * sizeof(T), alignment);

    // Total size is the max of the two main stages, plus persistent metadata/handler buffers.
    size_t total_bytes = persistent_buffers + std::max(decoder_layer_peak, final_stage_peak);

    // Part 2: Index and Metadata Vectors
    total_bytes += align_up(max_num_tokens * sizeof(uint32_t), alignment);
    total_bytes += align_up(max_num_tokens * sizeof(int32_t), alignment);
    total_bytes += align_up(max_num_tokens * sizeof(int32_t), alignment);
    total_bytes += align_up((max_batch_size + 1) * sizeof(int32_t), alignment);
    total_bytes += align_up(max_batch_size * sizeof(int32_t), alignment);
    total_bytes += align_up((max_batch_size + 1) * sizeof(int32_t), alignment);
    size_t max_mask_elements = max_num_tokens * max_kv_seqlens;
    total_bytes += align_up((max_mask_elements + 7) / 8, alignment);
    total_bytes += align_up((max_batch_size + 1) * sizeof(int32_t), alignment);
    total_bytes += align_up(max_num_tokens * sizeof(int32_t), alignment);
    total_bytes += align_up(max_num_tokens * sizeof(int32_t), alignment);

    return total_bytes;
}

template <typename T>
void L4maBuffer<T>::plan(
    hipStream_t strm,
     std::vector<int32_t>& input_ids_host,
     std::vector<int32_t>& position_ids_host,
     std::vector<int32_t>& kv_page_indices_host,
     std::vector<int32_t>& kv_page_indptr_host,
     std::vector<int32_t>& kv_last_page_lens_host,
     std::vector<int32_t>& qo_indptr_host,
     std::vector<bool>& custom_masks_host,
     std::vector<int32_t>& mask_indptr_host,
     std::vector<int32_t>& kv_batch_indices_host,
     std::vector<int32_t>& kv_positions_host,
     std::vector<int32_t>& output_indices_src_host
) {
    this->stream = strm;
    this->num_tokens = input_ids_host.size();
    this->batch_size = kv_page_indptr_host.empty() ? 0 : kv_page_indptr_host.size() - 1;

    std::vector<uint8_t> packed_custom_mask_host = packbits_little(custom_masks_host);

    allocator_->reset();

    input_ids          = allocator_->allocate_and_copy_async<int32_t>(input_ids_host, stream);
    position_ids       = allocator_->allocate_and_copy_async<int32_t>(position_ids_host, stream);
    kv_page_indices    = allocator_->allocate_and_copy_async<int32_t>(kv_page_indices_host, stream);
    kv_page_indptr     = allocator_->allocate_and_copy_async<int32_t>(kv_page_indptr_host, stream);
    kv_last_page_lens  = allocator_->allocate_and_copy_async<int32_t>(kv_last_page_lens_host, stream);
    qo_indptr          = allocator_->allocate_and_copy_async<int32_t>(qo_indptr_host, stream);
    custom_mask        = allocator_->allocate_and_copy_async<uint8_t>(packed_custom_mask_host, stream);
    mask_indptr        = allocator_->allocate_and_copy_async<int32_t>(mask_indptr_host, stream);
    kv_batch_indices   = allocator_->allocate_and_copy_async<int32_t>(kv_batch_indices_host, stream);
    kv_positions       = allocator_->allocate_and_copy_async<int32_t>(kv_positions_host, stream);

    if (!output_indices_src_host.empty()) {
        output_indices_src = allocator_->allocate_and_copy_async<int32_t>(output_indices_src_host, stream);
    }

    Tensor<uint8_t> flashinfer_float_buffer = this->allocate<uint8_t>(256 * 1024 * 1024);
    Tensor<uint8_t> flashinfer_int_buffer = this->allocate<uint8_t>(8 * 1024 * 1024);

    prefill_handler.Plan<T, int32_t>(
        flashinfer_float_buffer.data(), flashinfer_float_buffer.size(),
        flashinfer_int_buffer.data(), flashinfer_int_buffer.size(),
        qo_indptr_host.data(),
        kv_page_indptr_host.data(),
        num_tokens,
        batch_size,
        config.num_query_heads,
        config.num_key_value_heads,
        config.head_size,
        page_size
    );

}

template <typename T> template <typename U>
Tensor<U> L4maBuffer<T>::allocate(size_t count) {
    return allocator_->template allocate<U>(count);
}

template <typename T>
Tensor<uint8_t> L4maBuffer<T>::allocate_rest() {
    return allocator_->allocate_rest();
}

template <typename T> template <typename U>
void L4maBuffer<T>::deallocate(Tensor<U>& tensor) {
    allocator_->deallocate(tensor);
}


/// KV cache

template <typename T>
size_t L4maKVCache<T>::get_workspace_size(const L4maConfig& config, int32_t num_kv_pages, int32_t page_size) {
    size_t single_layer_elements = (size_t)num_kv_pages * page_size * config.num_key_value_heads * config.head_size;
    size_t all_layers_elements = config.num_layers * single_layer_elements;
    // Return size in bytes for both K and V caches
    return 2 * all_layers_elements * sizeof(T);
}

template <typename T>
L4maKVCache<T>::L4maKVCache(const L4maConfig& config, int32_t num_kv_pages, int32_t page_size)
    : config_(config), num_kv_pages_(num_kv_pages), page_size_(page_size) {
    size_t single_layer_elements = (size_t)num_kv_pages * page_size * config.num_key_value_heads * config.head_size;
    size_t total_elements = 2 * (size_t)config.num_layers * single_layer_elements;
    kv_cache_ = Tensor<T>(total_elements);
}

template <typename T>
std::pair<T*, T*> L4maKVCache<T>::get_layer_pointers(size_t layer_idx) {
    size_t layer_cache_size_elements = (size_t)num_kv_pages_ * page_size_ * config_.num_key_value_heads * config_.head_size;
    size_t total_k_cache_elements = (size_t)config_.num_layers * layer_cache_size_elements;

    T* k_base_ptr = kv_cache_.data();
    T* v_base_ptr = kv_cache_.data() + total_k_cache_elements;

    T* layer_k_ptr = k_base_ptr + layer_idx * layer_cache_size_elements;
    T* layer_v_ptr = v_base_ptr + layer_idx * layer_cache_size_elements;

    return {layer_k_ptr, layer_v_ptr};
}




// --- Constructor Implementations (Unchanged) ---
template <typename T>
RMSNorm<T>::RMSNorm(const L4maConfig& config)
    : config_(config), weight_(Tensor<T>(config.hidden_size)) {}

template <typename T>
L4maMlp<T>::L4maMlp(const L4maConfig& config)
    : config_(config),
      gate_proj_weights_(Tensor<T>(config.hidden_size * config.intermediate_size)),
      up_proj_weights_(Tensor<T>(config.hidden_size * config.intermediate_size)),
      down_proj_weights_(Tensor<T>(config.intermediate_size * config.hidden_size)) {}

template <typename T>
L4maAttention<T>::L4maAttention(const L4maConfig& config)
    : config_(config),
      q_proj_weights_(Tensor<T>(config.hidden_size * (config.num_query_heads * config.head_size))),
      k_proj_weights_(Tensor<T>(config.hidden_size * (config.num_key_value_heads * config.head_size))),
      v_proj_weights_(Tensor<T>(config.hidden_size * (config.num_key_value_heads * config.head_size))),
      o_proj_weights_(Tensor<T>((config.num_query_heads * config.head_size) * config.hidden_size)) {
    // if (config_.use_qkv_bias) {
    //     q_proj_bias_ = Tensor<T>(config.num_query_heads * config.head_size);
    //     k_proj_bias_ = Tensor<T>(config.num_key_value_heads * config.head_size);
    //     v_proj_bias_ = Tensor<T>(config.num_key_value_heads * config.head_size);
    // }
}

template <typename T>
L4maDecoderLayer<T>::L4maDecoderLayer(const L4maConfig& config)
    : config_(config),
      self_attn_(config),
      mlp_(config),
      input_layernorm_(config),
      post_attention_layernorm_(config) {}

template <typename T>
L4maModel<T>::L4maModel(const L4maConfig& config)
    : config_(config),
      embed_tokens_weight_(Tensor<T>(config.vocab_size * config.hidden_size)),
      norm_(config) {

    layers_.reserve(config.num_layers);
    for (int i = 0; i < config.num_layers; ++i) {
        layers_.emplace_back(config);
    }
}

template <typename T>
L4maForCausalLM<T>::L4maForCausalLM(const L4maConfig& config)
    : config_(config),
      model_(config) {

}

// --- KV Cache and Workspace Management (REFACTORED) ---

// template <typename T>
// void L4maForCausalLM<T>::create_kv_device_vectors(int max_kv_num) {
//     size_t kv_cache_size = static_cast<size_t>(max_kv_num) * config_.num_key_value_heads * config_.head_size * config_.num_layers;
//     if (kv_cache_k_.size() != kv_cache_size) {
//         kv_cache_k_.resize(kv_cache_size);
//     }
//     if (kv_cache_v_.size() != kv_cache_size) {
//         kv_cache_v_.resize(kv_cache_size);
//     }
// }


// --- get_parameters() Implementations (Corrected) ---
template <typename T>
std::map<std::string, Tensor<T>*> RMSNorm<T>::get_parameters() {
    // Return a pointer to the weight tensor
    return {{"weight", &weight_}};
}

template <typename T>
std::map<std::string, Tensor<T>*> L4maMlp<T>::get_parameters() {
    // Return pointers to the weight tensors
    return {{"gate_proj.weight", &gate_proj_weights_},
            {"up_proj.weight", &up_proj_weights_},
            {"down_proj.weight", &down_proj_weights_}};
}

template <typename T>
std::map<std::string, Tensor<T>*> L4maAttention<T>::get_parameters() {
    // Initialize the map with pointers
    auto params = std::map<std::string, Tensor<T>*>{
        {"q_proj.weight", &q_proj_weights_},
        {"k_proj.weight", &k_proj_weights_},
        {"v_proj.weight", &v_proj_weights_},
        {"o_proj.weight", &o_proj_weights_}};
    // Bias handling (if you re-enable it)
    // if (config_.use_qkv_bias) {
    //     params["q_proj.bias"] = &q_proj_bias_;
    //     params["k_proj.bias"] = &k_proj_bias_;
    //     params["v_proj.bias"] = &v_proj_bias_;
    // }
    return params;
}

template <typename T>
std::map<std::string, Tensor<T>*> L4maDecoderLayer<T>::get_parameters() {
    // The map now correctly stores pointers
    std::map<std::string, Tensor<T>*> params;
    // The 'val' from the sub-calls is now a Tensor<T>*, which can be assigned directly.
    for (auto const& [key, val] : self_attn_.get_parameters()) { params["self_attn." + key] = val; }
    for (auto const& [key, val] : mlp_.get_parameters()) { params["mlp." + key] = val; }
    for (auto const& [key, val] : input_layernorm_.get_parameters()) { params["input_layernorm." + key] = val; }
    for (auto const& [key, val] : post_attention_layernorm_.get_parameters()) { params["post_attention_layernorm." + key] = val; }
    return params;
}

template <typename T>
std::map<std::string, Tensor<T>*> L4maModel<T>::get_parameters() {
    std::map<std::string, Tensor<T>*> params;
    params["embed_tokens.weight"] = &embed_tokens_weight_;
    for (size_t i = 0; i < layers_.size(); ++i) {
        for (auto const& [key, val] : layers_[i].get_parameters()) {
            params["layers." + std::to_string(i) + "." + key] = val;
        }
    }
    for (auto const& [key, val] : norm_.get_parameters()) { params["norm." + key] = val; }
    return params;
}

template <typename T>
std::map<std::string, Tensor<T>*> L4maForCausalLM<T>::get_parameters() {
    std::map<std::string, Tensor<T>*> params;
    for (auto const& [key, val] : model_.get_parameters()) {
        params["model." + key] = val;
    }

    return params;
}

template <typename T>
void RMSNorm<T>::forward(
    T* output,
    const T* input,
    int num_tokens,
    hipStream_t stream) {

    uint32_t d = config_.hidden_size;

    flashinfer::norm::RMSNorm<T>(
        const_cast<T *>(input),
        weight_.data(),
        output,
        num_tokens, d, d, d, config_.rms_norm_eps, false, stream
    );

    if (artifacts::op_enabled("rmsnorm_forward")) {
        std::string case_id = artifacts::get_env_str("PIE_ARTIFACT_CASE_ID", "auto");
        auto dir = artifacts::ensure_dir_for_case("rmsnorm_forward", case_id);
        artifacts::write_device_bin(dir, "input", input, (size_t)num_tokens * d);
        artifacts::write_device_bin(dir, "weight", weight_.data(), (size_t)d);
        artifacts::write_device_bin(dir, "output", output, (size_t)num_tokens * d);
       // meta (no per-tensor checksum; raw .bin files are the contract)
       std::ostringstream meta;
       meta << "\"version\": \"1\",\n"
           << "\"op\": \"rmsnorm_forward\",\n"
           << "\"case_id\": " << artifacts::json_escape(case_id) << ",\n"
           << "\"config\": {\"rms_norm_eps\": " << config_.rms_norm_eps << ", \"hidden_size\": " << d << ", \"num_tokens\": " << num_tokens << "},\n"
           << "\"dtype_map\": {\"input\": \"bf16\", \"weight\": \"bf16\", \"output\": \"bf16\"},\n"
           << "\"shape_map\": {\"input\": [" << num_tokens << ", " << d << "], \"weight\": [" << d << "], \"output\": [" << num_tokens << ", " << d << "]}";
       artifacts::write_meta_json(dir, meta.str());
    }
}

template <typename T>
void L4maMlp<T>::forward(
    ProfileScope profiler,
    L4maBuffer<T>& buffer,
    T* output,
    const T* x
) {
    const int hidden_size = config_.hidden_size;
    const int intermediate_size = config_.intermediate_size;
    const size_t proj_count = (size_t)buffer.num_tokens * intermediate_size;

    Tensor<T> up_proj_out = buffer.template allocate<T>(proj_count);
    Tensor<T> gate_proj_out = buffer.template allocate<T>(proj_count);

    // Use a Tensor<uint8_t> for the raw byte buffer
    size_t cublas_workspace_size = 32 * 1024 * 1024;
    Tensor<uint8_t> cublas_workspace = buffer.template allocate<uint8_t>(cublas_workspace_size);

    // 2. Gate and Up projections. TODO: Fuse them into a single GEMM if possible
    gemm_cublasLt<T>(buffer.ltHandle, buffer.stream, x, up_proj_weights_.data(), nullptr, up_proj_out.data(), buffer.num_tokens, intermediate_size, hidden_size, cublas_workspace.data(), cublas_workspace_size, false, true);
    profiler.record("up_projection");
    gemm_cublasLt<T>(buffer.ltHandle, buffer.stream, x, gate_proj_weights_.data(), nullptr, gate_proj_out.data(), buffer.num_tokens, intermediate_size, hidden_size, cublas_workspace.data(), cublas_workspace_size, false, true);
    profiler.record("gate_projection");

    // 3. SwiGLU activation (gate * silu(up))
    // We can reuse the gate_proj_out_ptr buffer for the output of SwiGLU
    silu_and_mul<T>(
        up_proj_out.data(),
        gate_proj_out.data(),
        up_proj_out.data(),
        buffer.num_tokens,
        intermediate_size,
        buffer.stream
    );
    profiler.record("silu_and_mul");
    //std::cout << "SwiGLU output mean: " << up_proj_out.mean() << std::endl;

    // 4. Down projection
    gemm_cublasLt<T>(buffer.ltHandle, buffer.stream, up_proj_out.data(), down_proj_weights_.data(), nullptr, output, buffer.num_tokens, hidden_size, intermediate_size, cublas_workspace.data(), cublas_workspace_size, false, true);
    profiler.record("down_projection");

    // Optional dumps for MLP
    if (artifacts::op_enabled("mlp_forward")) {
        std::string case_id = artifacts::get_env_str("PIE_ARTIFACT_CASE_ID", "auto");
        auto dir = artifacts::ensure_dir_for_case("mlp_forward", case_id);
        artifacts::write_device_bin(dir, "input", x, (size_t)buffer.num_tokens * hidden_size);
        artifacts::write_device_bin(dir, "gate_proj_weight", gate_proj_weights_.data(), (size_t)hidden_size * intermediate_size);
        artifacts::write_device_bin(dir, "up_proj_weight", up_proj_weights_.data(), (size_t)hidden_size * intermediate_size);
        artifacts::write_device_bin(dir, "down_proj_weight", down_proj_weights_.data(), (size_t)intermediate_size * hidden_size);
        artifacts::write_device_bin(dir, "gate_proj_out", gate_proj_out.data(), proj_count);
        artifacts::write_device_bin(dir, "up_proj_out", up_proj_out.data(), proj_count);
        artifacts::write_device_bin(dir, "output", output, (size_t)buffer.num_tokens * hidden_size);
       std::ostringstream meta;
       meta << "\"version\": \"1\",\n"
           << "\"op\": \"mlp_forward\",\n"
           << "\"case_id\": " << artifacts::json_escape(case_id) << ",\n"
           << "\"config\": {\"hidden_size\": " << hidden_size << ", \"intermediate_size\": " << intermediate_size << ", \"num_tokens\": " << buffer.num_tokens << "},\n"
           << "\"dtype_map\": {\"input\": \"bf16\", \"gate_proj_weight\": \"bf16\", \"up_proj_weight\": \"bf16\", \"down_proj_weight\": \"bf16\", \"gate_proj_out\": \"bf16\", \"up_proj_out\": \"bf16\", \"output\": \"bf16\"},\n"
           << "\"shape_map\": {\"input\": [" << buffer.num_tokens << ", " << hidden_size << "], \"gate_proj_weight\": [" << hidden_size << ", " << intermediate_size << "], \"up_proj_weight\": [" << hidden_size << ", " << intermediate_size << "], \"down_proj_weight\": [" << intermediate_size << ", " << hidden_size << "], \"gate_proj_out\": [" << buffer.num_tokens << ", " << intermediate_size << "], \"up_proj_out\": [" << buffer.num_tokens << ", " << intermediate_size << "], \"output\": [" << buffer.num_tokens << ", " << hidden_size << "]}";
       artifacts::write_meta_json(dir, meta.str());
    }

    // 5. Deallocate buffers in reverse order of allocation (LIFO)
    buffer.deallocate(cublas_workspace);
    buffer.deallocate(gate_proj_out);
    buffer.deallocate(up_proj_out);

}

template <typename T>
void L4maAttention<T>::forward(
    ProfileScope profiler,
    L4maBuffer<T>& buffer,
    T* attn_output,
    const T* hidden_states,
    T* kv_cache_k,
    T* kv_cache_v
) {

    const size_t num_tokens = buffer.num_tokens;
    const size_t hidden_size = config_.hidden_size;
    const size_t head_size = config_.head_size;
    const size_t num_query_heads = config_.num_query_heads;
    const size_t num_key_value_heads = config_.num_key_value_heads;
    const size_t batch_size = buffer.batch_size;

    const size_t q_proj_count = (size_t)num_tokens * num_query_heads * head_size;
    const size_t kv_proj_count = (size_t)num_tokens * num_key_value_heads * head_size;

    // 1. Allocate buffers from the stack allocator
    Tensor<T> q_proj = buffer.template allocate<T>(q_proj_count);
    Tensor<T> k_proj = buffer.template allocate<T>(kv_proj_count);
    Tensor<T> v_proj = buffer.template allocate<T>(kv_proj_count);
    size_t cublas_workspace_size = 32 * 1024 * 1024;
    Tensor<uint8_t> cublas_workspace = buffer.template allocate<uint8_t>(cublas_workspace_size);

    // 2. Q, K, V projections. TODO: Fuse them into a single GEMM if possible
    gemm_cublasLt<T>(buffer.ltHandle, buffer.stream, hidden_states, q_proj_weights_.data(), nullptr, q_proj.data(), num_tokens, num_query_heads * head_size, hidden_size, cublas_workspace.data(), cublas_workspace_size, false, true);
    profiler.record("q_projection");
    gemm_cublasLt<T>(buffer.ltHandle, buffer.stream, hidden_states, k_proj_weights_.data(), nullptr, k_proj.data(), num_tokens, num_key_value_heads * head_size, hidden_size, cublas_workspace.data(), cublas_workspace_size, false, true);
    profiler.record("k_projection");
    gemm_cublasLt<T>(buffer.ltHandle, buffer.stream, hidden_states, v_proj_weights_.data(), nullptr, v_proj.data(), num_tokens, num_key_value_heads * head_size, hidden_size, cublas_workspace.data(), cublas_workspace_size, false, true);
    profiler.record("v_projection");

    // Optional: dump pre-RoPE projections and inputs
    if (artifacts::op_enabled("l4ma_attention_forward")) {
        std::string case_id = artifacts::get_env_str("PIE_ARTIFACT_CASE_ID", "auto");
        auto dir = artifacts::ensure_dir_for_case("l4ma_attention_forward", case_id);
        // Minimal meta; richer fields can be added later
        std::ostringstream meta;
        meta << "\"version\": \"1\",\n"
             << "\"op\": \"l4ma_attention_forward\",\n"
             << "\"case_id\": " << artifacts::json_escape(case_id) << ",\n"
             << "\"config\": {\"hidden_size\": " << hidden_size
             << ", \"head_size\": " << head_size
             << ", \"num_q_heads\": " << num_query_heads
             << ", \"num_kv_heads\": " << num_key_value_heads
             << ", \"page_size\": " << buffer.page_size
             << ", \"batch_size\": " << batch_size
           << ", \"num_tokens\": " << num_tokens << "},\n";
       // Dtype map (all bf16 here except index tensors which are s32/u8)
       meta << "\"dtype_map\": {"
           << "\"hidden_states\": \"bf16\","
           << "\"q_proj\": \"bf16\", \"k_proj\": \"bf16\", \"v_proj\": \"bf16\","
           << "\"q_after_rope\": \"bf16\", \"k_after_rope\": \"bf16\","
           << "\"context_before_o_proj\": \"bf16\", \"attn_output\": \"bf16\","
           << "\"q_proj_weight\": \"bf16\", \"k_proj_weight\": \"bf16\", \"v_proj_weight\": \"bf16\", \"o_proj_weight\": \"bf16\","
           << "\"position_ids\": \"s32\", \"kv_page_indices\": \"s32\", \"kv_page_indptr\": \"s32\", \"kv_last_page_lens\": \"s32\","
           << "\"kv_batch_indices\": \"s32\", \"kv_positions\": \"s32\", \"qo_indptr\": \"s32\", \"custom_mask\": \"u8\", \"mask_indptr\": \"s32\"},\n";
       // Shape map (flattened row-major tensors)
       meta << "\"shape_map\": {";
       meta << "\"hidden_states\": [" << num_tokens << ", " << hidden_size << "],";
       meta << "\"q_proj\": [" << num_tokens << ", " << (num_query_heads * head_size) << "],";
       meta << "\"k_proj\": [" << num_tokens << ", " << (num_key_value_heads * head_size) << "],";
       meta << "\"v_proj\": [" << num_tokens << ", " << (num_key_value_heads * head_size) << "],";
       meta << "\"q_after_rope\": [" << num_tokens << ", " << (num_query_heads * head_size) << "],";
       meta << "\"k_after_rope\": [" << num_tokens << ", " << (num_key_value_heads * head_size) << "],";
       meta << "\"context_before_o_proj\": [" << num_tokens << ", " << (num_query_heads * head_size) << "],";
       meta << "\"attn_output\": [" << num_tokens << ", " << hidden_size << "],";
       // Weight shapes: (in_dim, out_dim) in row-major flattened order
       meta << "\"q_proj_weight\": [" << hidden_size << ", " << (num_query_heads * head_size) << "],";
       meta << "\"k_proj_weight\": [" << hidden_size << ", " << (num_key_value_heads * head_size) << "],";
       meta << "\"v_proj_weight\": [" << hidden_size << ", " << (num_key_value_heads * head_size) << "],";
       meta << "\"o_proj_weight\": [" << (num_query_heads * head_size) << ", " << hidden_size << "],";
       meta << "\"position_ids\": [" << num_tokens << "],";
       meta << "\"kv_page_indices\": [" << buffer.kv_page_indices.size() << "],";
       meta << "\"kv_page_indptr\": [" << buffer.kv_page_indptr.size() << "],";
       meta << "\"kv_last_page_lens\": [" << buffer.kv_last_page_lens.size() << "],";
       meta << "\"kv_batch_indices\": [" << buffer.kv_batch_indices.size() << "],";
       meta << "\"kv_positions\": [" << buffer.kv_positions.size() << "],";
       meta << "\"qo_indptr\": [" << buffer.qo_indptr.size() << "],";
       meta << "\"custom_mask\": [" << buffer.custom_mask.size() << "],";
       meta << "\"mask_indptr\": [" << buffer.mask_indptr.size() << "]};\n";
        artifacts::write_meta_json(dir, meta.str());

        // Host-side metadata/index arrays
        artifacts::write_device_bin(dir, "hidden_states", hidden_states, num_tokens * hidden_size);
       // Projection weights (allowing Metal to recompute q/k/v if desired)
       artifacts::write_device_bin(dir, "q_proj_weight", q_proj_weights_.data(), (size_t)hidden_size * num_query_heads * head_size);
       artifacts::write_device_bin(dir, "k_proj_weight", k_proj_weights_.data(), (size_t)hidden_size * num_key_value_heads * head_size);
       artifacts::write_device_bin(dir, "v_proj_weight", v_proj_weights_.data(), (size_t)hidden_size * num_key_value_heads * head_size);
       artifacts::write_device_bin(dir, "o_proj_weight", o_proj_weights_.data(), (size_t)num_query_heads * head_size * hidden_size);
        artifacts::write_device_bin(dir, "q_proj", q_proj.data(), q_proj_count);
        artifacts::write_device_bin(dir, "k_proj", k_proj.data(), kv_proj_count);
        artifacts::write_device_bin(dir, "v_proj", v_proj.data(), kv_proj_count);
        artifacts::write_device_bin(dir, "position_ids", buffer.position_ids.data(), num_tokens);
        artifacts::write_device_bin(dir, "kv_page_indices", buffer.kv_page_indices.data(), buffer.kv_page_indices.size());
        artifacts::write_device_bin(dir, "kv_page_indptr", buffer.kv_page_indptr.data(), buffer.kv_page_indptr.size());
        artifacts::write_device_bin(dir, "kv_last_page_lens", buffer.kv_last_page_lens.data(), buffer.kv_last_page_lens.size());
        artifacts::write_device_bin(dir, "kv_batch_indices", buffer.kv_batch_indices.data(), buffer.kv_batch_indices.size());
        artifacts::write_device_bin(dir, "kv_positions", buffer.kv_positions.data(), buffer.kv_positions.size());
        artifacts::write_device_bin(dir, "qo_indptr", buffer.qo_indptr.data(), buffer.qo_indptr.size());
        artifacts::write_device_bin(dir, "custom_mask", buffer.custom_mask.data(), buffer.custom_mask.size());
        artifacts::write_device_bin(dir, "mask_indptr", buffer.mask_indptr.data(), buffer.mask_indptr.size());
    }

    flashinfer::paged_kv_t<T, int32_t> paged_kv(
        num_key_value_heads, buffer.page_size, head_size, batch_size,
        flashinfer::QKVLayout::kNHD,
        kv_cache_k, kv_cache_v,
        buffer.kv_page_indices.data(),
        buffer.kv_page_indptr.data(),
        buffer.kv_last_page_lens.data()
    );
    profiler.record("kv_page_create");

    // 3. Apply RoPE (in-place)
    hipError_t status = flashinfer::BatchQKApplyLlama31RotaryPosIds(
        q_proj.data(), k_proj.data(), q_proj.data(),  k_proj.data(),
        buffer.position_ids.data(),
        (uint32_t)num_tokens, (uint32_t)num_query_heads, (uint32_t)num_key_value_heads, (uint32_t)head_size, (uint32_t)head_size,
        num_query_heads * head_size, head_size, num_key_value_heads * head_size, head_size,
        num_query_heads * head_size, head_size, num_key_value_heads * head_size, head_size,
        false, config_.rope_factor, config_.rope_theta, config_.rope_low_frequency_factor,
        config_.rope_high_frequency_factor, 8192, buffer.stream
    );

    profiler.record("apply_rope");

    // Optional: dump q/k after RoPE application
    if (artifacts::op_enabled("l4ma_attention_forward")) {
        std::string case_id = artifacts::get_env_str("PIE_ARTIFACT_CASE_ID", "auto");
        auto dir = artifacts::ensure_dir_for_case("l4ma_attention_forward", case_id);
    // Ensure RoPE kernel has completed so we capture post-RoPE contents
    hipStreamSynchronize(buffer.stream);
        artifacts::write_device_bin(dir, "q_after_rope", q_proj.data(), q_proj_count);
        artifacts::write_device_bin(dir, "k_after_rope", k_proj.data(), kv_proj_count);
    }

    flashinfer::AppendPagedKVCache<T, int32_t>(
        paged_kv, k_proj.data(), v_proj.data(),
        buffer.kv_batch_indices.data(),
        buffer.kv_positions.data(),
        num_tokens,
        num_key_value_heads * head_size, head_size,
        num_key_value_heads * head_size, head_size,
        buffer.stream
    );
    profiler.record("append_kv_cache");

    // Reuse a buffer for the attention output before the final projection
    T* o_proj_input_ptr = q_proj.data();
    flashinfer::BatchPrefillWithPagedKVCacheWrapper<T, T, T, int32_t>(
        &buffer.prefill_handler, q_proj.data(), buffer.qo_indptr.data(),
        nullptr, paged_kv, o_proj_input_ptr, nullptr, num_query_heads,
        flashinfer::MaskMode::kCustom,
        buffer.custom_mask.data(),
        buffer.mask_indptr.data(),
        flashinfer::PosEncodingMode::kNone,
        false, // use_fp16_qk_reduction -> unused
        std::nullopt, // maybe_sm_scale -> unused
        1.f, // rope_scale -> unused
        1e4, // rope_theta -> unused
        buffer.stream
    );
    profiler.record("attention");

    // Optional: dump attention context before o-proj
    if (artifacts::op_enabled("l4ma_attention_forward")) {
        std::string case_id = artifacts::get_env_str("PIE_ARTIFACT_CASE_ID", "auto");
        auto dir = artifacts::ensure_dir_for_case("l4ma_attention_forward", case_id);
        artifacts::write_device_bin(dir, "context_before_o_proj", o_proj_input_ptr, num_tokens * num_query_heads * head_size);
    }

    // 5. Final output projection
    gemm_cublasLt<T>(buffer.ltHandle, buffer.stream, o_proj_input_ptr, o_proj_weights_.data(), nullptr, attn_output, num_tokens, hidden_size, num_query_heads * head_size, cublas_workspace.data(), cublas_workspace_size, false, true);
    profiler.record("o_projection");

    // Optional: dump final attention output post o-proj
    if (artifacts::op_enabled("l4ma_attention_forward")) {
        std::string case_id = artifacts::get_env_str("PIE_ARTIFACT_CASE_ID", "auto");
        auto dir = artifacts::ensure_dir_for_case("l4ma_attention_forward", case_id);
        artifacts::write_device_bin(dir, "attn_output", attn_output, num_tokens * hidden_size);
    // Overwrite meta.json with a comprehensive schema (no checksum_map).
    hipStreamSynchronize(buffer.stream);
    std::ostringstream meta_full;
    meta_full << "\"version\": \"1\",\n"
          << "\"op\": \"l4ma_attention_forward\",\n"
          << "\"case_id\": " << artifacts::json_escape(case_id) << ",\n"
          << "\"config\": {\"hidden_size\": " << hidden_size
          << ", \"head_size\": " << head_size
          << ", \"num_q_heads\": " << num_query_heads
          << ", \"num_kv_heads\": " << num_key_value_heads
          << ", \"page_size\": " << buffer.page_size
          << ", \"batch_size\": " << batch_size
          << ", \"num_tokens\": " << num_tokens
          << ", \"rope_theta\": " << config_.rope_theta
          << ", \"rope_factor\": " << config_.rope_factor
          << ", \"rope_low_freq_factor\": " << config_.rope_low_frequency_factor
          << ", \"rope_high_freq_factor\": " << config_.rope_high_frequency_factor << "},\n"
          << "\"dtype_map\": {"
          << "\"hidden_states\": \"bf16\",\"q_proj\": \"bf16\",\"k_proj\": \"bf16\",\"v_proj\": \"bf16\"," 
          << "\"q_after_rope\": \"bf16\",\"k_after_rope\": \"bf16\",\"context_before_o_proj\": \"bf16\",\"attn_output\": \"bf16\","
          << "\"q_proj_weight\": \"bf16\",\"k_proj_weight\": \"bf16\",\"v_proj_weight\": \"bf16\",\"o_proj_weight\": \"bf16\","
          << "\"position_ids\": \"s32\",\"kv_page_indices\": \"s32\",\"kv_page_indptr\": \"s32\",\"kv_last_page_lens\": \"s32\"," 
          << "\"kv_batch_indices\": \"s32\",\"kv_positions\": \"s32\",\"qo_indptr\": \"s32\",\"custom_mask\": \"u8\",\"mask_indptr\": \"s32\"},\n"
          << "\"shape_map\": {"
          << "\"hidden_states\": [" << num_tokens << ", " << hidden_size << "],"
          << "\"q_proj\": [" << num_tokens << ", " << (num_query_heads * head_size) << "],"
          << "\"k_proj\": [" << num_tokens << ", " << (num_key_value_heads * head_size) << "],"
          << "\"v_proj\": [" << num_tokens << ", " << (num_key_value_heads * head_size) << "],"
          << "\"q_after_rope\": [" << num_tokens << ", " << (num_query_heads * head_size) << "],"
          << "\"k_after_rope\": [" << num_tokens << ", " << (num_key_value_heads * head_size) << "],"
          << "\"context_before_o_proj\": [" << num_tokens << ", " << (num_query_heads * head_size) << "],"
          << "\"attn_output\": [" << num_tokens << ", " << hidden_size << "],"
          << "\"q_proj_weight\": [" << hidden_size << ", " << (num_query_heads * head_size) << "],"
          << "\"k_proj_weight\": [" << hidden_size << ", " << (num_key_value_heads * head_size) << "],"
          << "\"v_proj_weight\": [" << hidden_size << ", " << (num_key_value_heads * head_size) << "],"
          << "\"o_proj_weight\": [" << (num_query_heads * head_size) << ", " << hidden_size << "],"
          << "\"position_ids\": [" << num_tokens << "],"
          << "\"kv_page_indices\": [" << buffer.kv_page_indices.size() << "],"
          << "\"kv_page_indptr\": [" << buffer.kv_page_indptr.size() << "],"
          << "\"kv_last_page_lens\": [" << buffer.kv_last_page_lens.size() << "],"
          << "\"kv_batch_indices\": [" << buffer.kv_batch_indices.size() << "],"
          << "\"kv_positions\": [" << buffer.kv_positions.size() << "],"
          << "\"qo_indptr\": [" << buffer.qo_indptr.size() << "],"
          << "\"custom_mask\": [" << buffer.custom_mask.size() << "],"
          << "\"mask_indptr\": [" << buffer.mask_indptr.size() << "]}";
    artifacts::write_meta_json(dir, meta_full.str());
    }

    // 6. Deallocate buffers in reverse order
    buffer.deallocate(cublas_workspace);
    buffer.deallocate(v_proj);
    buffer.deallocate(k_proj);
    buffer.deallocate(q_proj);
}

template <typename T>
void L4maDecoderLayer<T>::forward(
    ProfileScope profiler,
    L4maBuffer<T>& buffer,
    T* hidden_states,
    T* kv_cache_k,
    T* kv_cache_v
) {
    const int num_tokens = buffer.num_tokens;
    const size_t hidden_size_elements = (size_t)num_tokens * config_.hidden_size;

    // --- 1. Self-Attention Block ---
    // The input `hidden_states` serves as the first residual.
    Tensor<T> normed_input = buffer.template allocate<T>(hidden_size_elements);
    input_layernorm_.forward(normed_input.data(), hidden_states, num_tokens, buffer.stream);
    profiler.record("norm_1");

    Tensor<T> attn_output = buffer.template allocate<T>(hidden_size_elements);

    self_attn_.forward(profiler.scope("self_attn"), buffer, attn_output.data(),
                       normed_input.data() , kv_cache_k, kv_cache_v);

    //logger.record("self_attn", buffer.stream);


    add_residual_kernel<<<(hidden_size_elements + 255) / 256, 256, 0, buffer.stream>>>(
        hidden_states, attn_output.data(), hidden_size_elements);
    profiler.record("attn_residual_add");

    // Deallocate attn_output and then normed_input to free up space for the MLP block
    buffer.deallocate(attn_output);
    buffer.deallocate(normed_input);


    // --- 2. MLP Block ---
    // The result of the attention block, `hidden_states`, is the residual for the MLP block.
    Tensor<T> normed_mlp_input = buffer.template allocate<T>(hidden_size_elements);
    post_attention_layernorm_.forward(normed_mlp_input.data(), hidden_states, num_tokens, buffer.stream);
    profiler.record("norm_2");

    Tensor<T> mlp_output = buffer.template allocate<T>(hidden_size_elements);
    mlp_.forward(profiler.scope("mlp"), buffer, mlp_output.data(), normed_mlp_input.data());
    // print the attn_output_ptr mean for debugging
    // float attn_output_mean = compute_mean(mlp_output_ptr, hidden_size_elements);
    // std::cout << "mlp_output_ptr mean: " << attn_output_mean << std::endl;

    add_residual_kernel<<<(hidden_size_elements + 255) / 256, 256, 0, buffer.stream>>>(
        hidden_states, mlp_output.data(), hidden_size_elements);
    profiler.record("mlp_residual_add");

    // Deallocate MLP buffers
    buffer.deallocate(mlp_output);
    buffer.deallocate(normed_mlp_input);
}

template <typename T>
void L4maModel<T>::forward(
    ProfileScope profiler,
    L4maBuffer<T>& buffer,
    L4maKVCache<T>& kv_cache,
    T* final_norm_output
) {
    const int num_tokens = buffer.num_tokens;
    const size_t hidden_size_elements = (size_t)num_tokens * config_.hidden_size;

    // Allocate a working buffer for the layers. The layers will operate in-place on this buffer.
    Tensor<T> working_hidden_buffer = buffer.template allocate<T>(hidden_size_elements);

    embed<T, int32_t>(
        embed_tokens_weight_.data(),
        embed_tokens_weight_.size() / config_.hidden_size,
        buffer.input_ids.data(),
        buffer.num_tokens,
        working_hidden_buffer.data(), // Embeddings are written to the allocated working buffer
        config_.hidden_size,
        buffer.stream
    );
    profiler.record("embedding_lookup");

    // print out the mean of the embeddings
    // float embed_mean = compute_mean(working_hidden_buffer, hidden_size_elements);
    // std::cout << "Embed mean: " << embed_mean << std::endl;

    for (size_t i = 0; i < layers_.size(); ++i) {

        auto& layer = layers_[i];

        auto [layer_k_cache_ptr, layer_v_cache_ptr] = kv_cache.get_layer_pointers(i);

        layer.forward(profiler.scope("decoder_layer"), buffer, working_hidden_buffer.data(),
                      layer_k_cache_ptr, layer_v_cache_ptr);

    }

    // Final norm reads from the working buffer and writes to the final output buffer.
    norm_.forward(final_norm_output, working_hidden_buffer.data(), num_tokens, buffer.stream);
    profiler.record("norm_");

    // Deallocate the working buffer.
    buffer.deallocate(working_hidden_buffer);
}

template <typename T>
std::pair<std::vector<float>, std::vector<int32_t>> L4maForCausalLM<T>::forward(
    ProfileScope profiler,
    L4maBuffer<T>& buffer,
    L4maKVCache<T>& kv_cache
) {

    const int num_tokens = buffer.num_tokens;
    const int num_output_tokens = buffer.output_indices_src.size();
    const int dist_size = buffer.dist_size;
    const size_t hidden_elements = (size_t)num_tokens * config_.hidden_size;
    const size_t output_elements = (size_t)num_output_tokens * config_.hidden_size;
    const size_t lm_head_workspace_bytes = 32 * 1024 * 1024;

    // 1. Allocate all necessary temporary buffers from the stack allocator.
    Tensor<T> hidden_states = buffer.template allocate<T>(hidden_elements);

    model_.forward(
        profiler.scope("model"),
        buffer,
        kv_cache,
        hidden_states.data()
    );

    if (num_output_tokens == 0) {
        // If there are no output tokens, we can return empty vectors.
        return std::make_pair(std::vector<float>(), std::vector<int32_t>());
    }


    Tensor<T> output_logits = buffer.template allocate<T>(num_output_tokens * config_.vocab_size);
    Tensor<float> output_logits_fp32 = buffer.template allocate<float>(num_output_tokens * config_.vocab_size);
    Tensor<float> output_logits_masked = buffer.template allocate<float>(num_output_tokens * config_.vocab_size);
    Tensor<float> final_logits_val = buffer.template allocate<float>(num_output_tokens * dist_size);
    Tensor<int32_t> final_logits_indices = buffer.template allocate<int32_t>(num_output_tokens * dist_size);
    Tensor<uint8_t> lm_head_workspace = buffer.template allocate<uint8_t>(lm_head_workspace_bytes);



    // 3. Handle the hidden states for the final projection.
    Tensor<T>* final_hidden_states_ptr = &hidden_states;
    Tensor<T> gathered_states;
    bool needs_gather = (hidden_elements != output_elements);

    if (needs_gather) {
        // NOTE: gathered_states is allocated last, so it must be deallocated first.
        gathered_states = buffer.template allocate<T>(output_elements);
        embed<T, int32_t>(
            hidden_states.data(),
            num_tokens,
            buffer.output_indices_src.data(),
            num_output_tokens,
            gathered_states.data(),
            config_.hidden_size,
            buffer.stream
        );
        final_hidden_states_ptr = &gathered_states;
        profiler.record("gather_hidden_states");

    }

    // 4. Compute logits
    gemm_cublasLt<T>(
        buffer.ltHandle, buffer.stream,
        final_hidden_states_ptr->data(),
        model_.get_embed_tokens_weight().data(),
        nullptr,
        output_logits.data(),
        num_output_tokens,
        config_.vocab_size, config_.hidden_size,
        lm_head_workspace.data(), lm_head_workspace_bytes, false, true
    );
    profiler.record("lm_head");

    cast_type<T, float>(
        output_logits.data(),
        output_logits_fp32.data(),
        num_output_tokens * config_.vocab_size,
        buffer.stream
    );
    profiler.record("casting");

    // 5. Perform sampling
    flashinfer::sampling::TopKMaskLogits<float, int32_t>(
        output_logits_fp32.data(),
        output_logits_masked.data(),
        nullptr,
        num_output_tokens,
        dist_size,
        config_.vocab_size,
        buffer.stream
    );
    profiler.record("topkmask");

    extract_k_values<float>(
        output_logits_masked.data(),
        final_logits_val.data(),
        final_logits_indices.data(),
        num_output_tokens,
        config_.vocab_size,
        dist_size,
        buffer.stream
    );
    profiler.record("extract");

    // 6. Copy final results back to the host.
    std::vector<float> final_logits_val_host = final_logits_val.to_vector();
    std::vector<int32_t> final_logits_indices_host = final_logits_indices.to_vector();

    // 7. DEALLOCATE ALL BUFFERS IN REVERSE ORDER (LIFO)


    if (needs_gather) {
        buffer.deallocate(gathered_states);
    }

    buffer.deallocate(lm_head_workspace);
    buffer.deallocate(final_logits_indices);
    buffer.deallocate(final_logits_val);
    buffer.deallocate(output_logits_masked);
    buffer.deallocate(output_logits_fp32); // Deallocate the fp32 version
    buffer.deallocate(output_logits);
    buffer.deallocate(hidden_states);

    // 8. Return the results.
    return std::make_pair(final_logits_val_host, final_logits_indices_host);
}

// --- Explicit Template Instantiations (Unchanged) ---
template class RMSNorm<__hip_bfloat16>;
template class L4maKVCache<__hip_bfloat16>;
template class L4maBuffer<__hip_bfloat16>;
template class L4maMlp<__hip_bfloat16>;
template class L4maAttention<__hip_bfloat16>;
template class L4maDecoderLayer<__hip_bfloat16>;
template class L4maModel<__hip_bfloat16>;
template class L4maForCausalLM<__hip_bfloat16>;
